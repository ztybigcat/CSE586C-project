#include "hip/hip_runtime.h"
// gpu_radix_sorter.cu
#include "gpu_radix_sorter.h"

__device__ __forceinline__ int getDigit(int key, int bitOffset)
{
    return (key >> bitOffset) & (RADIX - 1);
}

__global__ void histogramKernel(const int *keys, int *histograms, int n, int bitOffset)
{
    __shared__ int localHist[RADIX];

    // Initialize local histogram
    for (int i = threadIdx.x; i < RADIX; i += blockDim.x)
    {
        localHist[i] = 0;
    }
    __syncthreads();

    // Each block handles its portion
    int itemsPerBlock = (n + gridDim.x - 1) / gridDim.x;
    int blockStart = blockIdx.x * itemsPerBlock;
    int blockEnd = min(blockStart + itemsPerBlock, n);

    // Count elements in this block's range
    for (int i = blockStart + threadIdx.x; i < blockEnd; i += blockDim.x)
    {
        int digit = getDigit(keys[i], bitOffset);
        atomicAdd(&localHist[digit], 1);
    }
    __syncthreads();

    // Write local histogram to global memory
    for (int i = threadIdx.x; i < RADIX; i += blockDim.x)
    {
        atomicAdd(&histograms[i], localHist[i]);
    }
}

__global__ void reorderKernel(const int *keys_in, const int *values_in, int *keys_out, int *values_out,
                              const int *baseOffsets, const int *digitCounts, int *currentOffsets,
                              int n, int bitOffset, int *errorFlag)
{

    // Each block handles its portion
    int itemsPerBlock = (n + gridDim.x - 1) / gridDim.x;
    int blockStart = blockIdx.x * itemsPerBlock;
    int blockEnd = min(blockStart + itemsPerBlock, n);

    // Each thread in the block handles one element at a time
    for (int i = blockStart; i < blockEnd; i++)
    {
        if (threadIdx.x == (i - blockStart) % blockDim.x)
        {
            int key = keys_in[i];
            int value = values_in[i];
            int digit = getDigit(key, bitOffset);

            // Get exclusive position in output array
            int writePos = baseOffsets[digit] + atomicAdd(&currentOffsets[digit], 1);

            if (writePos < baseOffsets[digit] + digitCounts[digit])
            {
                atomicExch(&keys_out[writePos], key);
                atomicExch(&values_out[writePos], value);
            }
        }
        __syncthreads(); // Ensure one element is fully processed before moving to next
    }
}

void sortDataGPU_radix(const std::vector<int> &A, const std::vector<int> &B,
                       std::vector<int> &A_sorted, std::vector<int> &B_sorted)
{
    std::uint64_t N = A.size();
    if (N == 0)
        return;

    // Allocate device memory
    int *d_keys_in;
    int *d_values_in;
    int *d_keys_out;
    int *d_values_out;
    int *d_histograms;
    int *d_errorFlag;

    CUDA_CHECK(hipMalloc(&d_keys_in, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_values_in, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_keys_out, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_values_out, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_histograms, RADIX * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_errorFlag, sizeof(int)));

    // Copy data from host to device
    CUDA_CHECK(hipMemcpy(d_keys_in, B.data(), N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_values_in, A.data(), N * sizeof(int), hipMemcpyHostToDevice));

    // Calculate grid dimensions
    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    numBlocks = min(numBlocks, 1024);

    int numBits = sizeof(int) * 8;
    int numPasses = (numBits + RADIX_BITS - 1) / RADIX_BITS;

    for (int pass = 0; pass < numPasses; ++pass)
    {
        int bitOffset = pass * RADIX_BITS;

        // Reset histograms and error flag
        CUDA_CHECK(hipMemset(d_histograms, 0, RADIX * sizeof(int)));
        CUDA_CHECK(hipMemset(d_errorFlag, 0, sizeof(int)));

        // Step 1: Compute histograms
        histogramKernel<<<numBlocks, BLOCK_SIZE>>>(d_keys_in, d_histograms, N, bitOffset);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Step 2: Get histogram data from device
        std::vector<int> h_globalHistogram(RADIX);
        CUDA_CHECK(hipMemcpy(h_globalHistogram.data(), d_histograms,
                              RADIX * sizeof(int), hipMemcpyDeviceToHost));

        // Compute offsets
        std::vector<int> h_digitOffsets(RADIX);
        int total = 0;
        for (int i = 0; i < RADIX; ++i)
        {
            h_digitOffsets[i] = total;
            total += h_globalHistogram[i];
        }

        // Allocate and copy data for reorder kernel
        int *d_baseOffsets, *d_digitCounts, *d_currentOffsets;
        CUDA_CHECK(hipMalloc(&d_baseOffsets, RADIX * sizeof(int)));
        CUDA_CHECK(hipMalloc(&d_digitCounts, RADIX * sizeof(int)));
        CUDA_CHECK(hipMalloc(&d_currentOffsets, RADIX * sizeof(int)));

        CUDA_CHECK(hipMemcpy(d_baseOffsets, h_digitOffsets.data(),
                              RADIX * sizeof(int), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_digitCounts, h_globalHistogram.data(),
                              RADIX * sizeof(int), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemset(d_currentOffsets, 0, RADIX * sizeof(int)));

        // Step 3: Reorder elements
        reorderKernel<<<numBlocks, BLOCK_SIZE>>>(
            d_keys_in, d_values_in, d_keys_out, d_values_out,
            d_baseOffsets, d_digitCounts, d_currentOffsets,
            N, bitOffset, d_errorFlag);
        // After kernel launch:
        if (pass == 0)
        { // Only for first pass
            std::vector<int> check_keys(N);
            std::vector<int> check_values(N);
            CUDA_CHECK(hipMemcpy(check_keys.data(), d_keys_out, N * sizeof(int), hipMemcpyDeviceToHost));
            CUDA_CHECK(hipMemcpy(check_values.data(), d_values_out, N * sizeof(int), hipMemcpyDeviceToHost));
        }
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Check for errors
        int h_errorFlag;
        CUDA_CHECK(hipMemcpy(&h_errorFlag, d_errorFlag, sizeof(int), hipMemcpyDeviceToHost));
        if (h_errorFlag)
        {
            printf("Error detected in pass %d\n", pass);

            // Verify final offsets
            std::vector<int> final_offsets(RADIX);
            CUDA_CHECK(hipMemcpy(final_offsets.data(), d_currentOffsets,
                                  RADIX * sizeof(int), hipMemcpyDeviceToHost));

            printf("Final offset verification:\n");
            for (int i = 0; i < RADIX; i++)
            {
                if (final_offsets[i] != h_globalHistogram[i])
                {
                    printf("Digit %d: expected count %d, got %d\n",
                           i, h_globalHistogram[i], final_offsets[i]);
                }
            }
        }

        // Cleanup pass-specific allocations
        CUDA_CHECK(hipFree(d_baseOffsets));
        CUDA_CHECK(hipFree(d_digitCounts));
        CUDA_CHECK(hipFree(d_currentOffsets));

        // Swap buffers for next pass
        std::swap(d_keys_in, d_keys_out);
        std::swap(d_values_in, d_values_out);
    }

    // Copy final results back to host
    B_sorted.resize(N);
    A_sorted.resize(N);
    CUDA_CHECK(hipMemcpy(B_sorted.data(), d_keys_in, N * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(A_sorted.data(), d_values_in, N * sizeof(int), hipMemcpyDeviceToHost));

    // Cleanup
    CUDA_CHECK(hipFree(d_keys_in));
    CUDA_CHECK(hipFree(d_values_in));
    CUDA_CHECK(hipFree(d_keys_out));
    CUDA_CHECK(hipFree(d_values_out));
    CUDA_CHECK(hipFree(d_histograms));
    CUDA_CHECK(hipFree(d_errorFlag));
}