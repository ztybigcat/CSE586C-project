#include "hip/hip_runtime.h"
// gpu_radix_sorter.cu
#include "gpu_radix_sorter.h"

__device__ __forceinline__ int getDigit(int key, int bitOffset) {
    return (key >> bitOffset) & (RADIX - 1);
}

// Modified histogram kernel with proper synchronization
__global__ void histogramKernel(const int* keys, int* histograms, int n, int bitOffset) {
    __shared__ int localHist[RADIX];

    int tid = threadIdx.x;
    int block_start = blockIdx.x * blockDim.x;
    
    // Initialize local histogram
    for (int i = tid; i < RADIX; i += blockDim.x) {
        localHist[i] = 0;
    }
    __syncthreads();

    // Process multiple elements per thread if n > blockDim.x
    for (int idx = block_start + tid; idx < n; idx += blockDim.x * gridDim.x) {
        int key = keys[idx];
        int digit = getDigit(key, bitOffset);
        atomicAdd(&localHist[digit], 1);
    }
    __syncthreads();

    // Write back to global memory
    for (int i = tid; i < RADIX; i += blockDim.x) {
        histograms[blockIdx.x * RADIX + i] = localHist[i];
    }
}

// Modified reorder kernel with proper index handling
__global__ void reorderKernel(const int* keys_in, const int* values_in, int* keys_out, int* values_out,
                              const int* blockOffsets, int n, int bitOffset) {
    __shared__ int digitCounters[RADIX];
    
    // Initialize shared memory
    for (int i = threadIdx.x; i < RADIX; i += blockDim.x) {
        digitCounters[i] = 0;
    }
    __syncthreads();

    // Process multiple elements per thread if needed
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < n; idx += blockDim.x * gridDim.x) {
        int key = keys_in[idx];
        int value = values_in[idx];
        int digit = getDigit(key, bitOffset);

        // Get local position within this block for this digit
        int localPos = atomicAdd(&digitCounters[digit], 1);
        
        // Calculate global position
        int globalPos = blockOffsets[blockIdx.x * RADIX + digit] + localPos;
        
        // Write to output arrays
        if (globalPos < n) {  // Add bounds check
            keys_out[globalPos] = key;
            values_out[globalPos] = value;
        }
    }
}

void sortDataGPU_radix(const std::vector<int>& A, const std::vector<int>& B,
                       std::vector<int>& A_sorted, std::vector<int>& B_sorted) {
    std::uint64_t N = A.size();
    if (N == 0) return;

    // Allocate device memory
    int* d_keys_in;
    int* d_values_in;
    int* d_keys_out;
    int* d_values_out;

    CUDA_CHECK(hipMalloc(&d_keys_in, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_values_in, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_keys_out, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_values_out, N * sizeof(int)));

    // Copy data from host to device
    CUDA_CHECK(hipMemcpy(d_keys_in, B.data(), N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_values_in, A.data(), N * sizeof(int), hipMemcpyHostToDevice));

    // Calculate grid dimensions
    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    numBlocks = min(numBlocks, 1024);  // Limit number of blocks to avoid excessive memory usage
    
    // Allocate memory for histograms
    int* d_histograms;
    CUDA_CHECK(hipMalloc(&d_histograms, numBlocks * RADIX * sizeof(int)));

    int numBits = sizeof(int) * 8;
    int numPasses = (numBits + RADIX_BITS - 1) / RADIX_BITS;

    for (int pass = 0; pass < numPasses; ++pass) {
        int bitOffset = pass * RADIX_BITS;

        // Initialize histograms
        CUDA_CHECK(hipMemset(d_histograms, 0, numBlocks * RADIX * sizeof(int)));

        // Step 1: Compute local histograms
        histogramKernel<<<numBlocks, BLOCK_SIZE>>>(d_keys_in, d_histograms, N, bitOffset);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Step 2: Compute global histogram on CPU
        std::vector<int> h_histograms(numBlocks * RADIX);
        CUDA_CHECK(hipMemcpy(h_histograms.data(), d_histograms, numBlocks * RADIX * sizeof(int), hipMemcpyDeviceToHost));

        std::vector<int> h_globalHistogram(RADIX, 0);
        for (int i = 0; i < numBlocks; ++i) {
            for (int j = 0; j < RADIX; ++j) {
                h_globalHistogram[j] += h_histograms[i * RADIX + j];
            }
        }

        // Step 3: Compute exclusive prefix sum
        int total = 0;
        for (int i = 0; i < RADIX; ++i) {
            int temp = h_globalHistogram[i];
            h_globalHistogram[i] = total;
            total += temp;
        }

        // Compute block offsets
        std::vector<int> h_blockOffsets(numBlocks * RADIX);
        for (int d = 0; d < RADIX; ++d) {
            int sum = h_globalHistogram[d];
            for (int b = 0; b < numBlocks; ++b) {
                h_blockOffsets[b * RADIX + d] = sum;
                sum += h_histograms[b * RADIX + d];
            }
        }

        // Copy block offsets to device
        int* d_blockOffsets;
        CUDA_CHECK(hipMalloc(&d_blockOffsets, numBlocks * RADIX * sizeof(int)));
        CUDA_CHECK(hipMemcpy(d_blockOffsets, h_blockOffsets.data(), numBlocks * RADIX * sizeof(int), hipMemcpyHostToDevice));

        // Step 4: Reorder elements
        reorderKernel<<<numBlocks, BLOCK_SIZE>>>(d_keys_in, d_values_in, d_keys_out, d_values_out,
                                                 d_blockOffsets, N, bitOffset);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Swap input and output arrays for next pass
        std::swap(d_keys_in, d_keys_out);
        std::swap(d_values_in, d_values_out);

        CUDA_CHECK(hipFree(d_blockOffsets));
    }

    // Copy sorted data back to host
    B_sorted.resize(N);
    A_sorted.resize(N);
    CUDA_CHECK(hipMemcpy(B_sorted.data(), d_keys_in, N * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(A_sorted.data(), d_values_in, N * sizeof(int), hipMemcpyDeviceToHost));

    // Free device memory
    CUDA_CHECK(hipFree(d_keys_in));
    CUDA_CHECK(hipFree(d_values_in));
    CUDA_CHECK(hipFree(d_keys_out));
    CUDA_CHECK(hipFree(d_values_out));
    CUDA_CHECK(hipFree(d_histograms));
}