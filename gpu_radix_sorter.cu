#include "hip/hip_runtime.h"
// gpu_radix_sorter.cu
#include "gpu_radix_sorter.h"

// Function prototypes for kernels
__global__ void histogramKernel(const int* keys, int* histograms, int n, int bitOffset);
__global__ void reorderKernel(const int* keys_in, const int* values_in, int* keys_out, int* values_out,
                              const int* globalHistogram, int n, int bitOffset);

// Device function to get digit
__device__ __forceinline__ int getDigit(int key, int bitOffset) {
    return (key >> bitOffset) & (RADIX - 1);
}

// Radix Sort
void sortDataGPU_radix(const std::vector<int>& A, const std::vector<int>& B,
                       std::vector<int>& A_sorted, std::vector<int>& B_sorted) {
    std::uint64_t N = A.size();

    // Allocate device memory
    int* d_keys_in;
    int* d_values_in;
    int* d_keys_out;
    int* d_values_out;

    CUDA_CHECK(hipMalloc(&d_keys_in, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_values_in, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_keys_out, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_values_out, N * sizeof(int)));

    // Copy data from host to device
    CUDA_CHECK(hipMemcpy(d_keys_in, B.data(), N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_values_in, A.data(), N * sizeof(int), hipMemcpyHostToDevice));

    // Allocate memory for histograms
    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int* d_histograms;
    CUDA_CHECK(hipMalloc(&d_histograms, numBlocks * RADIX * sizeof(int)));

    // Radix sort parameters
    int numBits = sizeof(int) * 8; // 32 bits for int
    int numPasses = (numBits + RADIX_BITS - 1) / RADIX_BITS;

    for (int pass = 0; pass < numPasses; ++pass) {
        int bitOffset = pass * RADIX_BITS;

        // Initialize histograms
        CUDA_CHECK(hipMemset(d_histograms, 0, numBlocks * RADIX * sizeof(int)));

        // Step 1: Compute local histograms
        histogramKernel<<<numBlocks, BLOCK_SIZE>>>(d_keys_in, d_histograms, N, bitOffset);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Step 2: Compute global histogram (sum of local histograms)
        int* h_histograms = new int[numBlocks * RADIX];
        CUDA_CHECK(hipMemcpy(h_histograms, d_histograms, numBlocks * RADIX * sizeof(int), hipMemcpyDeviceToHost));

        int* h_globalHistogram = new int[RADIX];
        std::fill(h_globalHistogram, h_globalHistogram + RADIX, 0);

        for (int i = 0; i < numBlocks; ++i) {
            for (int j = 0; j < RADIX; ++j) {
                h_globalHistogram[j] += h_histograms[i * RADIX + j];
            }
        }

        // Step 3: Compute exclusive prefix sum (scan) on global histogram
        int sum = 0;
        for (int i = 0; i < RADIX; ++i) {
            int temp = h_globalHistogram[i];
            h_globalHistogram[i] = sum;
            sum += temp;
        }

        // Copy global histogram back to device
        int* d_globalHistogram;
        CUDA_CHECK(hipMalloc(&d_globalHistogram, RADIX * sizeof(int)));
        CUDA_CHECK(hipMemcpy(d_globalHistogram, h_globalHistogram, RADIX * sizeof(int), hipMemcpyHostToDevice));

        // Step 4: Reorder elements based on computed indices
        reorderKernel<<<numBlocks, BLOCK_SIZE>>>(d_keys_in, d_values_in, d_keys_out, d_values_out,
                                                 d_globalHistogram, N, bitOffset);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());

        // Swap input and output arrays for next pass
        std::swap(d_keys_in, d_keys_out);
        std::swap(d_values_in, d_values_out);

        // Clean up
        delete[] h_histograms;
        delete[] h_globalHistogram;
        CUDA_CHECK(hipFree(d_globalHistogram));
    }

    // Copy sorted data back to host
    B_sorted.resize(N);
    A_sorted.resize(N);
    CUDA_CHECK(hipMemcpy(B_sorted.data(), d_keys_in, N * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(A_sorted.data(), d_values_in, N * sizeof(int), hipMemcpyDeviceToHost));

    // Free device memory
    CUDA_CHECK(hipFree(d_keys_in));
    CUDA_CHECK(hipFree(d_values_in));
    CUDA_CHECK(hipFree(d_keys_out));
    CUDA_CHECK(hipFree(d_values_out));
    CUDA_CHECK(hipFree(d_histograms));
}

// Kernel to compute local histograms
__global__ void histogramKernel(const int* keys, int* histograms, int n, int bitOffset) {
    __shared__ int localHist[RADIX];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    // Initialize local histogram
    for (int i = tid; i < RADIX; i += blockDim.x) {
        localHist[i] = 0;
    }
    __syncthreads();

    // Accumulate local histogram
    if (idx < n) {
        int key = keys[idx];
        int digit = getDigit(key, bitOffset);
        atomicAdd(&localHist[digit], 1);
    }
    __syncthreads();

    // Write local histogram to global memory
    for (int i = tid; i < RADIX; i += blockDim.x) {
        histograms[blockIdx.x * RADIX + i] = localHist[i];
    }
}

// Kernel to reorder elements based on computed indices
__global__ void reorderKernel(const int* keys_in, const int* values_in, int* keys_out, int* values_out,
                              const int* globalHistogram, int n, int bitOffset) {
    __shared__ int localHist[RADIX];
    __shared__ int localScan[RADIX];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    // Initialize local histogram
    for (int i = tid; i < RADIX; i += blockDim.x) {
        localHist[i] = 0;
    }
    __syncthreads();

    // Build local histogram
    if (idx < n) {
        int key = keys_in[idx];
        int digit = getDigit(key, bitOffset);
        atomicAdd(&localHist[digit], 1);
    }
    __syncthreads();

    // Compute exclusive scan on local histogram
    if (tid < RADIX) {
        int sum = 0;
        for (int i = 0; i < tid; ++i) {
            sum += localHist[i];
        }
        localScan[tid] = sum;
    }
    __syncthreads();

    // Compute base address for each digit
    __shared__ int base[RADIX];
    if (tid < RADIX) {
        base[tid] = globalHistogram[tid];
    }
    __syncthreads();

    // Write elements to output array
    if (idx < n) {
        int key = keys_in[idx];
        int value = values_in[idx];
        int digit = getDigit(key, bitOffset);

        int pos = base[digit] + localScan[digit] + tid - localScan[digit];

        keys_out[pos] = key;
        values_out[pos] = value;

        atomicAdd(&base[digit], 1);
    }
}
