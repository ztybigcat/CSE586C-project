#include "hip/hip_runtime.h"
#include "gpu_custom_sorter.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdint>
#include <algorithm>
#include <cmath>
#include <climits> // For INT_MAX

// Error checking macro
#define CUDA_CHECK(call)                                                         \
    {                                                                            \
        hipError_t err = call;                                                  \
        if (err != hipSuccess) {                                                \
            std::cerr << "CUDA Error: " << hipGetErrorString(err)               \
                      << " at " << __FILE__ << ":" << __LINE__ << std::endl;     \
            exit(EXIT_FAILURE);                                                  \
        }                                                                        \
    }

// Define block size for kernels
#define BLOCK_SIZE 256

// Function prototypes
__global__ void bitonicSortKernel(int* d_keys, int* d_values, int N, int stage, int passOfStage);
__global__ void bitonicSortKernelShared(int* d_keys, int* d_values, int N, int stage, int passOfStage);

// Old GPU sorter using global memory
void sortDataGPU_custom_old(const std::vector<int>& A, const std::vector<int>& B,
                        std::vector<int>& A_sorted, std::vector<int>& B_sorted) {
    std::uint64_t N = A.size();

    // Find the next power of two
    int log2N = std::ceil(std::log2(N));
    std::uint64_t N_padded = 1 << log2N;

    // Create padded arrays
    std::vector<int> A_padded = A;
    std::vector<int> B_padded = B;

    // Pad the arrays with maximum integer values
    if (N_padded > N) {
        A_padded.resize(N_padded, INT_MAX);
        B_padded.resize(N_padded, INT_MAX);
    }

    // Allocate device memory
    int* d_keys;
    int* d_values;

    CUDA_CHECK(hipMalloc(&d_keys, N_padded * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_values, N_padded * sizeof(int)));

    // Copy data from host to device
    CUDA_CHECK(hipMemcpy(d_keys, B_padded.data(), N_padded * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_values, A_padded.data(), N_padded * sizeof(int), hipMemcpyHostToDevice));

    // Set up kernel dimensions
    int num_blocks = (N_padded + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 blocks(num_blocks);
    dim3 threads(BLOCK_SIZE);

    // Main bitonic sort loop
    int totalStages = log2N;
    for (int stage = 1; stage <= totalStages; ++stage) {
        for (int passOfStage = stage; passOfStage > 0; --passOfStage) {
            bitonicSortKernel<<<blocks, threads>>>(d_keys, d_values, N_padded, stage, passOfStage);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
        }
    }

    // Allocate host memory for sorted data
    B_sorted.resize(N_padded);
    A_sorted.resize(N_padded);

    // Copy sorted data back to host
    CUDA_CHECK(hipMemcpy(B_sorted.data(), d_keys, N_padded * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(A_sorted.data(), d_values, N_padded * sizeof(int), hipMemcpyDeviceToHost));

    // Remove padding
    B_sorted.resize(N);
    A_sorted.resize(N);

    // Free device memory
    CUDA_CHECK(hipFree(d_keys));
    CUDA_CHECK(hipFree(d_values));
}

// New GPU sorter with shared memory optimization
void sortDataGPU_custom(const std::vector<int>& A, const std::vector<int>& B,
                        std::vector<int>& A_sorted, std::vector<int>& B_sorted) {
    std::uint64_t N = A.size();

    // Find the next power of two
    int log2N = std::ceil(std::log2(N));
    std::uint64_t N_padded = 1 << log2N;

    // Create padded arrays
    std::vector<int> A_padded = A;
    std::vector<int> B_padded = B;

    // Pad the arrays with maximum integer values
    if (N_padded > N) {
        A_padded.resize(N_padded, INT_MAX);
        B_padded.resize(N_padded, INT_MAX);
    }

    // Allocate device memory
    int* d_keys;
    int* d_values;

    CUDA_CHECK(hipMalloc(&d_keys, N_padded * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_values, N_padded * sizeof(int)));

    // Copy data from host to device
    CUDA_CHECK(hipMemcpy(d_keys, B_padded.data(), N_padded * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_values, A_padded.data(), N_padded * sizeof(int), hipMemcpyHostToDevice));

    // Set up kernel dimensions
    int num_blocks = (N_padded + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 blocks(num_blocks);
    dim3 threads(BLOCK_SIZE);

    // Main bitonic sort loop
    int totalStages = log2N;
    for (int stage = 1; stage <= totalStages; ++stage) {
        for (int passOfStage = stage; passOfStage > 0; --passOfStage) {
            bitonicSortKernelShared<<<blocks, threads>>>(d_keys, d_values, N_padded, stage, passOfStage);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
        }
    }

    // Allocate host memory for sorted data
    B_sorted.resize(N_padded);
    A_sorted.resize(N_padded);

    // Copy sorted data back to host
    CUDA_CHECK(hipMemcpy(B_sorted.data(), d_keys, N_padded * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(A_sorted.data(), d_values, N_padded * sizeof(int), hipMemcpyDeviceToHost));

    // Remove padding
    B_sorted.resize(N);
    A_sorted.resize(N);

    // Free device memory
    CUDA_CHECK(hipFree(d_keys));
    CUDA_CHECK(hipFree(d_values));
}

// Original Bitonic Sort Kernel
__global__ void bitonicSortKernel(int* d_keys, int* d_values, int N, int stage, int passOfStage) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    unsigned int pairDistance = 1 << (passOfStage - 1);
    unsigned int blockWidth = 1 << stage;

    unsigned int left = idx;
    unsigned int right = idx ^ pairDistance;

    if (right > left) {
        // Determine the sorting direction
        bool ascending = ((idx / blockWidth) % 2) == 0;

        // Compare and swap
        if ((d_keys[left] > d_keys[right]) == ascending) {
            // Swap keys
            int temp_key = d_keys[left];
            d_keys[left] = d_keys[right];
            d_keys[right] = temp_key;

            // Swap values
            int temp_value = d_values[left];
            d_values[left] = d_values[right];
            d_values[right] = temp_value;
        }
    }
}

__global__ void bitonicSortKernelShared(int* d_keys, int* d_values, int N, int stage, int passOfStage) {
    // Shared memory for the keys and values
    __shared__ int shared_keys[BLOCK_SIZE];
    __shared__ int shared_values[BLOCK_SIZE];

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;  // Ensure the global index is within bounds

    // Load elements into shared memory (using thread index for shared memory access)
    shared_keys[threadIdx.x] = d_keys[idx];
    shared_values[threadIdx.x] = d_values[idx];
    __syncthreads();  // Ensure all threads have loaded their data

    // Debug: Print loaded values into shared memory for block 0 and thread 0
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        //printf("Initial load - Stage %d, Pass %d: shared_keys[0] = %d, shared_values[0] = %d\n", stage, passOfStage, shared_keys[0], shared_values[0]);
    }

    unsigned int pairDistance = 1 << (passOfStage - 1);
    unsigned int blockWidth = 1 << stage;

    // Calculate local thread indices for the current block
    unsigned int local_idx = threadIdx.x;
    unsigned int local_pair_idx = local_idx ^ pairDistance;  // Calculate pair index in shared memory

    // Ensure the pair index is within block size (to avoid out-of-bounds access in shared memory)
    if (local_pair_idx < BLOCK_SIZE) {
        // Determine the sorting direction (ascending or descending)
        bool ascending = ((idx / blockWidth) % 2) == 0;

        // Synchronize before comparison
        __syncthreads();

        // Debug: Before comparison and swap for thread 0 in block 0
        if (threadIdx.x == 0 && blockIdx.x == 0) {
            //printf("Before swap - Stage %d, Pass %d: shared_keys[0] = %d, shared_keys[1] = %d\n", stage, passOfStage, shared_keys[0], shared_keys[1]);
        }

        // Compare and swap within shared memory
        if ((shared_keys[local_idx] > shared_keys[local_pair_idx]) == ascending) {
            // Swap keys
            int temp_key = shared_keys[local_idx];
            shared_keys[local_idx] = shared_keys[local_pair_idx];
            shared_keys[local_pair_idx] = temp_key;

            // Swap values
            int temp_value = shared_values[local_idx];
            shared_values[local_idx] = shared_values[local_pair_idx];
            shared_values[local_pair_idx] = temp_value;
        }

        // Synchronize after comparison and swap
        __syncthreads();

        // Debug: After comparison and swap for thread 0 in block 0
        if (threadIdx.x == 0 && blockIdx.x == 0) {
            //printf("After swap - Stage %d, Pass %d: shared_keys[0] = %d, shared_keys[1] = %d\n", stage, passOfStage, shared_keys[0], shared_keys[1]);
        }
    }

    // Synchronize before writing back to global memory
    __syncthreads();

    // Write back the sorted keys and values to global memory
    d_keys[idx] = shared_keys[local_idx];
    d_values[idx] = shared_values[local_idx];

    // Debug: Write back to global memory for thread 0 in block 0
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        //printf("Writeback - Stage %d, Pass %d: d_keys[0] = %d, d_values[0] = %d\n", stage, passOfStage, d_keys[0], d_values[0]);
    }
}
