#include "hip/hip_runtime.h"
// gpu_bitonic_sorter.cu
#include "gpu_bitonic_sorter.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include <cstdint>
#include <algorithm>
#include <limits.h>

#define BLOCK_SIZE 1024 //

// Function prototypes for kernels
__global__ void bitonicSortKernel(int* d_keys, int* d_values, int N, int stage, int passOfStage);
__global__ void bitonicSortKernelShared(int* d_keys, int* d_values, int N, int stage, int passOfStage);
__global__ void bitonicSortKernelHybrid(int* d_keys, int* d_values, int N, int stage, int passOfStage);
// Bitonic Sort using global memory
void sortDataGPU_bitonic(const std::vector<int>& A, const std::vector<int>& B,
                         std::vector<int>& A_sorted, std::vector<int>& B_sorted) {
    std::uint64_t N = A.size();

    // Find the next power of two
    int log2N = std::ceil(std::log2(N));
    std::uint64_t N_padded = 1ULL << log2N;

    // Create padded arrays
    std::vector<int> A_padded = A;
    std::vector<int> B_padded = B;

    // Pad the arrays with maximum integer values
    if (N_padded > N) {
        A_padded.resize(N_padded, INT_MAX);
        B_padded.resize(N_padded, INT_MAX);
    }

    // Allocate device memory
    int* d_keys;
    int* d_values;
    hipMalloc(&d_keys, N_padded * sizeof(int));
    hipMalloc(&d_values, N_padded * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_keys, B_padded.data(), N_padded * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, A_padded.data(), N_padded * sizeof(int), hipMemcpyHostToDevice);

    // Set up kernel dimensions
    int num_blocks = (N_padded + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 blocks(num_blocks);
    dim3 threads(BLOCK_SIZE);

    // Main bitonic sort loop
    int totalStages = log2N;
    for (int stage = 1; stage <= totalStages; ++stage) {
        for (int passOfStage = stage; passOfStage > 0; --passOfStage) {
            bitonicSortKernel<<<blocks, threads>>>(d_keys, d_values, N_padded, stage, passOfStage);
            hipDeviceSynchronize();
        }
    }

    // Allocate host memory for sorted data
    B_sorted.resize(N_padded);
    A_sorted.resize(N_padded);

    // Copy sorted data back to host
    hipMemcpy(B_sorted.data(), d_keys, N_padded * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(A_sorted.data(), d_values, N_padded * sizeof(int), hipMemcpyDeviceToHost);

    // Remove padding
    B_sorted.resize(N);
    A_sorted.resize(N);

    // Free device memory
    hipFree(d_keys);
    hipFree(d_values);
}

// Bitonic Sort with shared memory optimization
void sortDataGPU_bitonic_shared_memory(const std::vector<int>& A, const std::vector<int>& B,
                                       std::vector<int>& A_sorted, std::vector<int>& B_sorted) {
    std::uint64_t N = A.size();

    // Find the next power of two
    int log2N = std::ceil(std::log2(N));
    std::uint64_t N_padded = 1ULL << log2N;

    // Create padded arrays
    std::vector<int> A_padded = A;
    std::vector<int> B_padded = B;

    // Pad the arrays with maximum integer values
    if (N_padded > N) {
        A_padded.resize(N_padded, INT_MAX);
        B_padded.resize(N_padded, INT_MAX);
    }

    // Allocate device memory
    int* d_keys;
    int* d_values;
    hipMalloc(&d_keys, N_padded * sizeof(int));
    hipMalloc(&d_values, N_padded * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_keys, B_padded.data(), N_padded * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, A_padded.data(), N_padded * sizeof(int), hipMemcpyHostToDevice);

    // Set up kernel dimensions
    int num_blocks = (N_padded + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 blocks(num_blocks);
    dim3 threads(BLOCK_SIZE);

    // Main bitonic sort loop
    int totalStages = log2N;
    for (int stage = 1; stage <= totalStages; ++stage) {
        int blockWidth = 1 << stage;

        for (int passOfStage = stage; passOfStage > 0; --passOfStage) {
            if (blockWidth <= BLOCK_SIZE) {
                bitonicSortKernelShared<<<blocks, threads>>>(d_keys, d_values, N_padded, stage, passOfStage);
            } 
            if (blockWidth > BLOCK_SIZE)
            
            {
              bitonicSortKernel<<<blocks, threads>>>(d_keys, d_values, N_padded, stage, passOfStage);
            }
            else
            {
              bitonicSortKernel<<<blocks, threads>>>(d_keys, d_values, N_padded, stage, passOfStage);
            }
            hipDeviceSynchronize();
        }
    }

    // Allocate host memory for sorted data
    B_sorted.resize(N_padded);
    A_sorted.resize(N_padded);

    // Copy sorted data back to host
    hipMemcpy(B_sorted.data(), d_keys, N_padded * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(A_sorted.data(), d_values, N_padded * sizeof(int), hipMemcpyDeviceToHost);

    // Remove padding
    B_sorted.resize(N);
    A_sorted.resize(N);

    // Free device memory
    hipFree(d_keys);
    hipFree(d_values);
}
void sortDataGPU_bitonic_Hybrid(const std::vector<int>& A, const std::vector<int>& B,
                                       std::vector<int>& A_sorted, std::vector<int>& B_sorted) {
    std::uint64_t N = A.size();

    // Find the next power of two
    int log2N = std::ceil(std::log2(N));
    std::uint64_t N_padded = 1ULL << log2N;

    // Create padded arrays
    std::vector<int> A_padded = A;
    std::vector<int> B_padded = B;

    // Pad the arrays with maximum integer values
    if (N_padded > N) {
        A_padded.resize(N_padded, INT_MAX);
        B_padded.resize(N_padded, INT_MAX);
    }

    // Allocate device memory
    int* d_keys;
    int* d_values;
    hipMalloc(&d_keys, N_padded * sizeof(int));
    hipMalloc(&d_values, N_padded * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_keys, B_padded.data(), N_padded * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, A_padded.data(), N_padded * sizeof(int), hipMemcpyHostToDevice);

    // Set up kernel dimensions
    int num_blocks = (N_padded + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 blocks(num_blocks);
    dim3 threads(BLOCK_SIZE);

    // Main bitonic sort loop
    int totalStages = log2N;
    for (int stage = 1; stage <= totalStages; ++stage) {
        int blockWidth = 1 << stage;

        for (int passOfStage = stage; passOfStage > 0; --passOfStage) {
            if (blockWidth <= BLOCK_SIZE) {
                bitonicSortKernelHybrid<<<blocks, threads>>>(d_keys, d_values, N_padded, stage, passOfStage);
            } 
            if (blockWidth > BLOCK_SIZE)
            
            {
              bitonicSortKernel<<<blocks, threads>>>(d_keys, d_values, N_padded, stage, passOfStage);
            }
            else
            {
              bitonicSortKernel<<<blocks, threads>>>(d_keys, d_values, N_padded, stage, passOfStage);
            }
            hipDeviceSynchronize();
        }
    }

    // Allocate host memory for sorted data
    B_sorted.resize(N_padded);
    A_sorted.resize(N_padded);

    // Copy sorted data back to host
    hipMemcpy(B_sorted.data(), d_keys, N_padded * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(A_sorted.data(), d_values, N_padded * sizeof(int), hipMemcpyDeviceToHost);

    // Remove padding
    B_sorted.resize(N);
    A_sorted.resize(N);

    // Free device memory
    hipFree(d_keys);
    hipFree(d_values);
}
// Bitonic Sort Kernel using global memory
__global__ void bitonicSortKernel(int* d_keys, int* d_values, int N, int stage, int passOfStage) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    unsigned int pairDistance = 1 << (passOfStage - 1);
    unsigned int blockWidth = 1 << stage;

    unsigned int left = idx;
    unsigned int right = idx ^ pairDistance;

    if (right > left) {
        bool ascending = ((idx / blockWidth) % 2) == 0;
        if ((d_keys[left] > d_keys[right]) == ascending) {
            int temp_key = d_keys[left];
            d_keys[left] = d_keys[right];
            d_keys[right] = temp_key;

            int temp_value = d_values[left];
            d_values[left] = d_values[right];
            d_values[right] = temp_value;
        }
    }
}

// Bitonic Sort Kernel with shared memory optimization
__global__ void bitonicSortKernelShared(int* d_keys, int* d_values, int N, int stage, int passOfStage) {
   // Shared memory for the keys and values
    __shared__ int shared_keys[BLOCK_SIZE];
    __shared__ int shared_values[BLOCK_SIZE];

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;  // Ensure the global index is within bounds

    // Load elements into shared memory (using thread index for shared memory access)
    shared_keys[threadIdx.x] = d_keys[idx];
    shared_values[threadIdx.x] = d_values[idx];
    __syncthreads();  // Ensure all threads have loaded their data

    // Debug: Print loaded values into shared memory for block 0 and thread 0
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        //printf("Initial load - Stage %d, Pass %d: shared_keys[0] = %d, shared_values[0] = %d\n", stage, passOfStage, shared_keys[0], shared_values[0]);
    }

    unsigned int pairDistance = 1 << (passOfStage - 1);
    unsigned int blockWidth = 1 << stage;

    // Calculate local thread indices for the current block
    unsigned int local_idx = threadIdx.x;
    unsigned int local_pair_idx = local_idx ^ pairDistance;  // Calculate pair index in shared memory
    // Ensure the pair index is within block size (to avoid out-of-bounds access in shared memory)
    if (local_pair_idx < BLOCK_SIZE) {
        // Determine the sorting direction (ascending or descending)
        bool ascending = ((idx / blockWidth) % 2) == 0;

        // Synchronize before comparison
        __syncthreads();

        // Debug: Before comparison and swap for thread 0 in block 0
        if (threadIdx.x == 0 && blockIdx.x == 0) {
            //printf("Before swap - Stage %d, Pass %d: shared_keys[0] = %d, shared_keys[1] = %d\n", stage, passOfStage, shared_keys[0], shared_keys[1]);
        }

        // Compare and swap within shared memory
        if ((shared_keys[local_idx] > shared_keys[local_pair_idx]) == ascending) {
            // Swap keys
            int temp_key = shared_keys[local_idx];
            shared_keys[local_idx] = shared_keys[local_pair_idx];
            shared_keys[local_pair_idx] = temp_key;

            // Swap values
            int temp_value = shared_values[local_idx];
            shared_values[local_idx] = shared_values[local_pair_idx];
            shared_values[local_pair_idx] = temp_value;
        }

        // Synchronize after comparison and swap
        __syncthreads();
        if (threadIdx.x == 0 && blockIdx.x == 0) {
            //printf("After swap - Stage %d, Pass %d: shared_keys[0] = %d, shared_keys[1] = %d\n", stage, passOfStage, shared_keys[0], shared_keys[1]);
        }
}
}

__global__ void bitonicSortKernelHybrid(int* d_keys, int* d_values, int N, int stage, int passOfStage) {
    __shared__ int shared_keys[BLOCK_SIZE];
    __shared__ int shared_values[BLOCK_SIZE];

    unsigned int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int local_idx = threadIdx.x;

    if (global_idx >= N) return;

    unsigned int pairDistance = 1 << (passOfStage - 1);
    unsigned int blockWidth = 1 << stage;

    // If the stage is within block boundaries, use shared memory
    if (blockWidth <= BLOCK_SIZE) {
        // Load elements into shared memory
        shared_keys[local_idx] = d_keys[global_idx];
        shared_values[local_idx] = d_values[global_idx];
        __syncthreads();

        unsigned int local_pair_idx = local_idx ^ pairDistance;

        // Ensure the pair index is within shared memory bounds
        if (local_pair_idx < BLOCK_SIZE) {
            // Determine sorting direction (ascending or descending)
            bool ascending = ((global_idx / blockWidth) % 2) == 0;

            // Compare and swap within shared memory
            if ((shared_keys[local_idx] > shared_keys[local_pair_idx]) == ascending) {
                // Swap keys
                int temp_key = shared_keys[local_idx];
                shared_keys[local_idx] = shared_keys[local_pair_idx];
                shared_keys[local_pair_idx] = temp_key;

                // Swap values
                int temp_value = shared_values[local_idx];
                shared_values[local_idx] = shared_values[local_pair_idx];
                shared_values[local_pair_idx] = temp_value;
            }
        }

        // Synchronize before writing back to global memory
        __syncthreads();

        // Write sorted values back to global memory
        d_keys[global_idx] = shared_keys[local_idx];
        d_values[global_idx] = shared_values[local_idx];
    } 
    // If the stage spans across blocks, work directly in global memory
    else {
        unsigned int global_pair_idx = global_idx ^ pairDistance;
        
        // Ensure the pair index is within bounds of the entire array
        if (global_pair_idx < N) {
            // Determine sorting direction (ascending or descending)
            bool ascending = ((global_idx / blockWidth) % 2) == 0;

            // Perform comparison and swap in global memory
            if ((d_keys[global_idx] > d_keys[global_pair_idx]) == ascending) {
                // Swap keys
                int temp_key = d_keys[global_idx];
                d_keys[global_idx] = d_keys[global_pair_idx];
                d_keys[global_pair_idx] = temp_key;

                // Swap values
                int temp_value = d_values[global_idx];
                d_values[global_idx] = d_values[global_pair_idx];
                d_values[global_pair_idx] = temp_value;
            }
        }
    }
}

